
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void bw_test() {
    printf("Testing BW on block %d, thread %d!\n", 
           blockIdx.x, threadIdx.x);
}

__global__ void run_bw_test() {
    bw_test();
}

// C wrapper function implementation
extern "C" void launch_bw_test() {
    run_bw_test<<<1, 1>>>();
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", 
                hipGetErrorString(err));
        return;
    }
    
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", 
                hipGetErrorString(err));
        return;
    }
}
