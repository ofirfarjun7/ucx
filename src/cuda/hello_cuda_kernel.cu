#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hello_cuda.h"

__device__ void hello_kernel() {
    printf("Hello World from block %d, thread %d!\n", 
           blockIdx.x, threadIdx.x);
}

__global__ void run_hello() {
    hello_kernel();
}

// C wrapper function implementation
extern "C" void launch_hello() {
    run_hello<<<1, 1>>>();
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", 
                hipGetErrorString(err));
        return;
    }
    
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", 
                hipGetErrorString(err));
        return;
    }
}
